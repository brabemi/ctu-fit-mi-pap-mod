#include "hip/hip_runtime.h"
#define PARALLEL_OPENMP //< define for OpenMP, undefine for sequential
#include <omp.h> // OpenMP library

// if the following variable is NOT defined, program will not use any function nor include the CImg library
// this also means that other libraries are needed
//~ #define CIMG_VISUAL

//~ #define SSE_SQRT

#define LOGGING // all logging output except for the line with "#THREADS #SECONDS" info
#undef LOGGING

#ifdef CIMG_VISUAL
#include "CImg/CImg.h" // lib for visualisation
#endif
#include <cstdio> // printf
#include <cstdlib> // srand
#include <cmath> // sqrt
#include <xmmintrin.h> //SSERsqrt

// include our functions/structs
#include "generator/ioproc.h" // process input file
#include "generator/SimConfig.h" // Struct with simulation config/settings

// include CUDA libs
#include <hip/hip_runtime.h>
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime_api.h>


#define __CUDA_INTERNAL_COMPILATION__
#include "math_functions.h" // rsqrt
#undef __CUDA_INTERNAL_COMPILATION__

// current mapping of simulation parameters - just to clarify what is what 
#define WIDTH sconf.width // // x-coordinate, width of simulation area
#define HEIGHT sconf.height // y-coordinate, height of simulation area
#define DEPTH sconf.depth // z-coordinate, depth of simulation area
#define MAX_SPEED sconf.max_speed // maximal default speed of particle, eg. 4 => speed in (-4;4)
#define MAX_WEIGHT sconf.max_weight // maximal mass of particle, eg. 4 => speed in (0;4)
// amount of particles
#define AMOUNT sconf.amount // amount of particles
// number of simulation steps is specified in SimConfig struct as simulation_steps
#define SQRT_MAX_WEIGHT (int)sqrt(sconf.max_weight) // constant for calculation particle color {mass/SQRT_MAX_WEIGHT, 255, mass%SQRT_MAX_WEIGHT}
// ------------- END OF MODIFIED MAPPINGs

// parameters for calculations (movement of particles)
#define F_QUOC 0.0005 // compensatory quotient for  MAX_WEIGHT
#define BOUNCE_LOSS 0.8 // conversion rate of velocity on bounce, eg. 0.8 => 80% of speed after bounce

/*
CURRENT STATE:
- finite simulation of NBody
- bouncing, borders
- no particle collisions
- no detection if 

LAST UPDATE (just to clarify what happened - move to CURRENT STATE after review):
- inner loop in simulation is now parallelized with parallel reduction of accelerations
- define maximum steps of simulation - viz SimConfig::simulation_steps
- input (definition of particles) from file/as parameters from comm. line - viz generator.cpp and ioproc.cpp
- input constants as parameters (WIDTH, MAX_SPEED, ...) - viz generator.cpp
- for CImg visualisation, variable "CIMG_VISUAL" must be defined - uncomment #define CIMG_VISUAL at the beginning of this file

- created standalone generator (in directory "generator")
- simulation should be run with: ./simulator input.txt
- valgrind says the program leaks a bit (6 block every time) - with --leak-check=full --show-leak-kinds=all, it seems that libgomp causes that

- reworked makefile (make compile, make clean)

TODO:
...

*/

#ifdef CIMG_VISUAL
using namespace cimg_library; // -> no need to use cimg_library::function()
#else
using namespace std;
#endif

__device__ __host__ bool bounce(float x, float y, float z, const float maxX, const float maxY, const float maxZ) {
	return (x < 0) || (maxX < x) || (y < 0) || (maxY < y) || (z < 0) || (maxZ < z);
}

__device__ __host__ float debounce_vel(float vel, float pos, int min, int max) {
	if((pos < min) || (max < pos)) {
		return -1*BOUNCE_LOSS*vel;
	}
	return BOUNCE_LOSS*vel;
}

__device__ __host__ float debounce_pos (float pos, int min, int max) {
	if(pos < min) {
		return -1 * pos;
	}
	if(pos > max) {
		return max - (pos - max);
	}
	return pos;
}

static void HandleError(hipError_t err, const char * file, int line){
	if (err != hipSuccess) {
		printf("%s in %s at line %d\n", hipGetErrorString(err), file, line);
		exit(EXIT_FAILURE);
	}
}
#define HANDLE_ERROR( err ) (HandleError(err, __FILE__, __LINE__))

__global__ void CopyCoordinatesKernel(float4 * sourceCoords, float4 * newCoords, int n)
{
	int gtid = blockIdx.x * blockDim.x + threadIdx.x;
	if (gtid >= n) return;

	float4 source = sourceCoords[gtid];
	float4 newCoordsVec = newCoords[gtid];

	source.x = newCoordsVec.x;
	source.y = newCoordsVec.y;
	source.z = newCoordsVec.z;
	// source.m is still the same

	sourceCoords[gtid] = source;
}

__global__ void CopyCoordinatesKernelSeq(float4 * sourceCoords, float4 * newCoords, int n)
{
	for (int i = 0; i < n; i++)
	{
		float4 source = sourceCoords[i];
		float4 newCoordsVec = newCoords[i];

		source.x = newCoordsVec.x;
		source.y = newCoordsVec.y;
		source.z = newCoordsVec.z;
		// source.m is still the same

		sourceCoords[i] = source;
	}
}

__device__ float3 perParticleAcceleration(float4 first, float4 second, float3 aXYZ, float eps)
{
	float3 dXYZ;
	dXYZ.x = first.x - second.x;
	dXYZ.y = first.y - second.y;
	dXYZ.z = first.z - second.z;

	float tmp_sum = dXYZ.x * dXYZ.x + dXYZ.y * dXYZ.y + dXYZ.z * dXYZ.z + eps;
	
	//float invr = 1/sqrtf(tmp_sum);
	float invr = rsqrtf(tmp_sum);

	float invr3 = invr*invr*invr;

	float f = F_QUOC * second.w * invr3;

	aXYZ.x += dXYZ.x * f;
	aXYZ.y += dXYZ.y * f;
	aXYZ.z += dXYZ.z * f;
	return aXYZ;
}

#define OBOC // which kind of work distribution is selected

__global__ void OneStepSimulation(float4 * sourceCoords, float4 * newCoords,
	float4 * velocities, float eps, float dt, 
	int n, int offset,
	const float maxX, const float maxY, const float maxZ)
{
	int globalThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (globalThreadIndex >= n) return;

	// shared memory - used to cache particles from global memory
	extern __shared__ float4 particlesInSM[];
	// pointer to array of particles in global memory 
	float4 * particlesGlobal = sourceCoords;
	// iterators
	int i = 0; // iterator for particles, (for each particle ... (from 1 .. n))
	int blok = 0; // iterator for current block
	float3 aXYZ = { 0.0f, 0.0f, 0.0f }; // acceleration, to be counted
	
	float4 threadVector = particlesGlobal[globalThreadIndex]; // vector of particle assigned to this thread

	// control output
	//if (globalThreadIndex == 1) printf("%f %f %f %f\n", threadVector.x, threadVector.y, threadVector.z, threadVector.w);

	// pre-declaration of variables
	float3 dXYZ; // only assigned to, should be OK
	float tmp_sum; // only assigned to, should be OK
	float invr, invr3; // only assigned to, should be OK
	float f; // only assigned to, should be OK

	int maxParticles = blockDim.x; // amount of particles % threadsPerBlock

#ifdef OBOC // one block per one processor
	if (blockIdx.x == gridDim.x) { maxParticles = n % (blockDim.x+1); /*if (maxParticles == 0) maxParticles == blockDim.x;*/ }
#endif

	// for each particle ...
	for (i = 0; i < n; i += offset) { // offset == threadsPerBlock specified in calling of the kernel

		int indexOfParticle = blok * blockDim.x + threadIdx.x; // index of particle to be stored in SM

		particlesInSM[threadIdx.x] = particlesGlobal[indexOfParticle]; // copy from global to shared memory, "cache"

		__syncthreads(); // wait for every thread so the SM is full

		// count current subblock
		// original condition: j < blockDim.x
		for (int j = 0; j < maxParticles; j++) { // !!! this might cause problems, will work ONLY if every thread in this block copied particle to SM !!!
			
			dXYZ.x = threadVector.x - particlesInSM[j].x;
			dXYZ.y = threadVector.y - particlesInSM[j].y;
			dXYZ.z = threadVector.z - particlesInSM[j].z;

			tmp_sum = dXYZ.x * dXYZ.x + dXYZ.y * dXYZ.y + dXYZ.z * dXYZ.z + eps;

			//float invr = 1/sqrtf(tmp_sum);
			invr = rsqrtf(tmp_sum);

			invr3 = invr*invr*invr;

			f = F_QUOC * particlesInSM[j].w * invr3;

			aXYZ.x += dXYZ.x * f;
			aXYZ.y += dXYZ.y * f;
			aXYZ.z += dXYZ.z * f;
			//aXYZ = perParticleAcceleration(threadVector, particlesInSM[j], aXYZ, eps);
		}
		
		__syncthreads(); // wait for every thread so the SM is free to be edited
		blok++;
	}
	// load velocity from global memory
	float4 vel = velocities[globalThreadIndex];
	// update velocity
	vel.x += dt*aXYZ.x; /* update velocity of particle "i" */
	vel.y += dt*aXYZ.y;
	vel.z += dt*aXYZ.z;
	// no change to .w
	
	// update position of vector
	float4 newVec = threadVector;
	newVec.x = threadVector.x + dt*vel.x + 0.5*dt*dt*aXYZ.x;
	newVec.y = threadVector.y + dt*vel.y + 0.5*dt*dt*aXYZ.y;
	newVec.z = threadVector.z + dt*vel.z + 0.5*dt*dt*aXYZ.z;
	// no change to .w == mass of particle

	// check bouncing
	if (bounce(newVec.x, newVec.y, newVec.z, maxX, maxY, maxZ)) {
		// update of particle velocity, change direction and value (BOUNCE_LOSS)
		vel.x = debounce_vel(vel.x, newVec.x, 0, maxX);
		vel.y = debounce_vel(vel.y, newVec.y, 0, maxY);
		vel.z = debounce_vel(vel.z, newVec.z, 0, maxZ);
		// update of particle position
		newVec.x = debounce_pos(newVec.x, 0, maxX);
		newVec.y = debounce_pos(newVec.y, 0, maxY);
		newVec.z = debounce_pos(newVec.z, 0, maxZ);
	}

	// store new velocities in global memory
	velocities[globalThreadIndex] = vel;

	// store updated position in global memory - array "(x/y/z)new"
	newCoords[globalThreadIndex] = newVec;
}

__global__ void GPUPrintParticles(float4 * particles, int n)
{
	for (int i = 0; i < n; i++)
	{
		float4 myPosition = particles[i];
		printf("%d. %f %f %f %f\n", i, myPosition.x, myPosition.y, myPosition.z, myPosition.w);
	}
}

int main(int argc, char** argv) {
	
	// simulation configuration
	SimConfig sconf;
	int threads;

	#ifdef CIMG_VISUAL
	bool graphics;
	#endif

	// process input
	if(argc == 4)
	{
		threads = atoi(argv[1]);
		#ifdef CIMG_VISUAL
		graphics = atoi(argv[2]) == 1;
		#endif
		#ifdef LOGGING
		printf("Processing input file %s\n", argv[3]);
		#endif
		processInputFile(argv[3], sconf);
	}
	else
	{
		printf("ERROR: Wrong/Missing parameters.\n");
		printf("Expect: \t%s THREADS GRAPHICS[0,1] INPUT_FILE\n", argv[0]);
		return 0;
	}
	#ifdef LOGGING
	printf("Input file processed.\n");	
	printf("Particles: %d\tSteps: %d\n", sconf.amount, sconf.simulation_steps);
	#endif
	
	// iterators
	int i, j;
	
	// pointers to arrays
	float * x = sconf.x;
	float * y = sconf.y;
	float * z = sconf.z;
	
	float * m = sconf.m;
	
	float * vx = sconf.vx;
	float * vy = sconf.vy;
	float * vz = sconf.vz;
#ifdef WITH_CPU
	float * xnew = new float[sconf.amount];
	float * ynew = new float[sconf.amount];
	float * znew = new float[sconf.amount];
#endif
	// create an array that is in form suitable for float4 == 1 particle
	float * hostParticles = new float[sconf.amount * 4]; // x y z m
	float * hostVelocities = new float[sconf.amount * 4]; // vx vy vz 0
	for (i = 0; i < (sconf.amount); i++)
	{
		int base = i * 4;
		hostParticles[base + 0] = x[i];
		hostParticles[base + 1] = y[i];
		hostParticles[base + 2] = z[i];
		hostParticles[base + 3] = m[i];

		hostVelocities[base + 0] = vx[i];
		hostVelocities[base + 1] = vy[i];
		hostVelocities[base + 2] = vz[i];
		hostVelocities[base + 3] = 0.0f;
	}
	// initialize arrays on GPU
	float * devParticles = NULL, * devParticlesNew = NULL;
	float * devVelocities = NULL;
	HANDLE_ERROR(hipMalloc((void**)&devParticles, (4 * sconf.amount * sizeof(float))));
	HANDLE_ERROR(hipMalloc((void**)&devParticlesNew, (4 * sconf.amount * sizeof(float))));
	HANDLE_ERROR(hipMalloc((void**)&devVelocities, (4 * sconf.amount * sizeof(float))));
	// copy content
	HANDLE_ERROR(hipMemcpy(devParticles, hostParticles, (4 * sconf.amount), hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devVelocities, hostVelocities, (4 * sconf.amount), hipMemcpyHostToDevice));

#ifdef WITH_CPU
	// variables used in computations
	float ax, ay, az;
	float dx, dy, dz;
	float invr, invr3;
	float f;
#endif
	// definitino of "n" - used in algorithm on site
	int n = sconf.amount;
	
	// constants for computing particle movement 
	float dt = 0.1f; // original value was 0.0001, that was too little for current values of particle parameters 
	float eps = 0.005f;
	
	#ifdef CIMG_VISUAL
	// image ~ "drawing panel"
	CImg<unsigned char> img;
	CImgDisplay main_disp;
	// colours
	const unsigned char red[] = { 255,0,0 }, green[] = { 0,255,0 }, blue[] = { 0,0,255 };

	if(graphics) {
		// image ~ "drawing panel"
		img = CImg<unsigned char> ( sconf.width, sconf.height,1,3);

		// initialization of window
		img.fill(0); //< fill img with black colour
	
		// draw all points
		for(i = 0; i < n; i++) {
			const unsigned char color[] = {(unsigned char) (m[i]/SQRT_MAX_WEIGHT), 255, (unsigned char) (((int) m[i])%SQRT_MAX_WEIGHT)};
			img.draw_point(x[i],y[i],color);
		}
		// create a Window (caption Playground) and fill it with image	
		main_disp = CImgDisplay(img,"Playground");
	}
	#endif

	unsigned steps = 0;

	// count amount of blocks and threads needed
	hipDeviceProp_t prop;
	HANDLE_ERROR(hipGetDeviceProperties(&prop, 0)); // get device properties

	int threadsPerBlock = 0, numOfBlocks = 0;


#ifdef OBOC // one block per one processor
	/* ----------------------------------------------------- */
	/* One block per processor */
	numOfBlocks = prop.multiProcessorCount;
	threadsPerBlock = ((sconf.amount) / numOfBlocks);
	if (threadsPerBlock > prop.maxThreadsPerBlock)
	{
		printf("NOPE NOPE NOPE");
	}
	/* ----------------------------------------------------- */
#endif

#ifdef WARP // one warp per one block - very slow, better to give more warps
	/* ----------------------------------------------------- */
	/* One block per processor */
	threadsPerBlock = prop.warpSize*4; // MUST be power of two
	numOfBlocks = ((sconf.amount - 1) / threadsPerBlock);
	numOfBlocks += 1;
	/* ----------------------------------------------------- */
#endif

#ifdef DEFAULT
	/* ----------------------------------------------------- */
	
	// One possible configuration of blocks and threads
	threadsPerBlock = (prop.maxThreadsPerBlock / 2); // MUST be power of 2
	// why only half of max - SM has limited amount of registers etc.
	// if limits reached -> kernel will not start (HANDLE_ERROR(startkernel<<<>>>()) will detect and print out this error)
	// /2 is just a working guess, may be optimized

	numOfBlocks = ((sconf.amount-1) / threadsPerBlock);
	numOfBlocks += 1; // amount = 1024, maxThreadsPerBlock = 1024 -> 2 blocks, both only half of max threads
	
	/* ----------------------------------------------------- */
#endif
	printf("----------------------------\n");
	printf("Amount of blocks: %d\n", numOfBlocks);
	printf("Amount of threads per block: %d\n", threadsPerBlock);

	// CUDA time measurement with events
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipDeviceSynchronize(); // just in case ...

	// Start CUDA recording
	hipEventRecord(start); 
	// Start OpenMP time measurement
	double t1 = omp_get_wtime();

//#define WITH_CPU

	#ifdef LOGGING
	printf("Starting simulation ...\n");
	#endif
	while (steps < sconf.simulation_steps) {

		// GPU simulation step
		OneStepSimulation << < numOfBlocks, threadsPerBlock, (threadsPerBlock*sizeof(float4)) >> > ((float4 *)devParticles, (float4 *)devParticlesNew,
			(float4 *)devVelocities, eps, dt, n, threadsPerBlock,
			sconf.width, sconf.height, sconf.depth);
		HANDLE_ERROR(hipPeekAtLastError());
		hipDeviceSynchronize(); // wait for Kernel to finish

		#ifdef WITH_CPU
		// CPU: compute new coordinates of all particles in parallel
		#ifdef PARALLEL_OPENMP
		#pragma omp parallel for num_threads(threads) private(ax,ay,az,dx,dy,dz,invr,invr3,f)
		#endif
		for (i = 0; i < n; i++) { /* Foreach particle "i" ... */
			ax = 0.0;
			ay = 0.0;
			az = 0.0;

			for (j = 0; j < n; j++) { /* Loop over all particles "j" */
				dx = x[j] - x[i];
				dy = y[j] - y[i];
				dz = z[j] - z[i];


				invr = 1.0 / sqrtf(dx*dx + dy*dy + dz*dz + eps);

				invr3 = invr*invr*invr;
				f = F_QUOC*m[j] * invr3;
				ax += f*dx; /* accumulate the acceleration from gravitational attraction */
				ay += f*dy;
				az += f*dz;
			}

			vx[i] += dt*ax; /* update velocity of particle "i" */
			vy[i] += dt*ay;
			vz[i] += dt*az;

			xnew[i] = x[i] + dt*vx[i] + 0.5*dt*dt*ax; /* update position of particle "i" */
			ynew[i] = y[i] + dt*vy[i] + 0.5*dt*dt*ay;
			znew[i] = z[i] + dt*vz[i] + 0.5*dt*dt*az;

			if (bounce(xnew[i], ynew[i], znew[i], sconf.width, sconf.height, sconf.depth)) {
				// update of particle velocity, change direction and value (BOUNCE_LOSS)
				vx[i] = debounce_vel(vx[i], xnew[i], 0, WIDTH);
				vy[i] = debounce_vel(vy[i], ynew[i], 0, HEIGHT);
				vz[i] = debounce_vel(vz[i], znew[i], 0, DEPTH);
				// update of particle position
				xnew[i] = debounce_pos(xnew[i], 0, WIDTH);
				ynew[i] = debounce_pos(ynew[i], 0, HEIGHT);
				znew[i] = debounce_pos(znew[i], 0, DEPTH);
			}
		}
		#endif




		#ifdef CIMG_VISUAL
		if(graphics) {
			if(main_disp.is_closed()) break;
			
			for(i = 0; i < n; i++) {
				const unsigned char color[] = {(unsigned char) (m[i]/SQRT_MAX_WEIGHT), 255, (unsigned char) (((int) m[i])%SQRT_MAX_WEIGHT)};
				img.draw_circle(x[i],y[i],1,color);
				//~ img.draw_circle(x[i],y[i],2,green);
			}
		}
		#endif

		// GPU copy
		//CopyCoordinatesKernelSeq << <1, 1 >> >((float4 *)devParticles, (float4 *)devParticlesNew, n);
		CopyCoordinatesKernel << < numOfBlocks, threadsPerBlock >> >
			((float4 *)devParticles, (float4 *)devParticlesNew, n);
		HANDLE_ERROR(hipPeekAtLastError());
		hipDeviceSynchronize(); // wait for kernel to finish
		
		// CPU copy
		#ifdef WITH_CPU
		#ifdef PARALLEL_OPENMP
		#pragma omp parallel for num_threads(threads)
		#endif
		for(i=0; i<n; i++) { /* copy updated positions back into original arrays */
		if( bounce(xnew[i], ynew[i], znew[i], sconf.width, sconf.height, sconf.depth) ) {
				#ifdef LOGGING
				printf("Particle %d out of borders (x, y, z) = (%0.3f, %0.3f, %0.3f)\n", i, xnew[i], ynew[i], znew[i]);
				#endif
			}
			x[i] = xnew[i];
			y[i] = ynew[i];
			z[i] = znew[i];
		}
		#endif

		#ifdef CIMG_VISUAL
		if(graphics) {
			// redraw the image and show it in the window
			if(steps%500 == 0) {
				img.fill(0); //< black background 
			}
	
			// draw all particles
			for(i = 0; i < n; i++) {
				//~ const unsigned char color[] = {(unsigned char) (m[i]/SQRT_MAX_WEIGHT), 255, (unsigned char) (((int) m[i])%SQRT_MAX_WEIGHT)};
				img.draw_circle(x[i],y[i],1,red);
				//~ img.draw_circle(x[i],y[i],2,green);
			}
			// display the image in window	
			img.display(main_disp);
			
			// wait for some time
			cimg::wait(20); // in milisec
		}
		#endif
		if(100*steps % sconf.simulation_steps == 0) {
			#ifdef LOGGING
			printf("%.1f%% completed\n", 100.0*steps/sconf.simulation_steps);
			#endif
		}
		steps++;
	}
	/*
	// copy end state of simulation from device to host	
	HANDLE_ERROR(hipMemcpy(hostParticles, devParticles, (4 * sconf.amount), hipMemcpyDeviceToHost));
	HANDLE_ERROR(hipPeekAtLastError());

	hipDeviceSynchronize();
	HANDLE_ERROR(hipPeekAtLastError());

	// output directly from GPU
	GPUPrintParticles << <1, 1 >> >((float4 *)devParticles, 10);
	// output of values on CPU
	for (int i = 0; i < 10; i++)
	{
		printf("%d. %f %f %f %f\t", i, hostParticles[4*i], hostParticles[4*i + 1], hostParticles[4*i + 2], hostParticles[4*i + 3]);
		printf("%d. %f %f %f %f\n", i, x[i], y[i], z[i], m[i]);
	}*/
	/*
    // Control - differences are quite large in case of many steps (500 and more)
	double diffTol = 100.0;
	for (int i = 0; i < n; i++)
	{
		int base = i * 4;
		if (m[i] != hostParticles[base + 3]) printf("Masses of particle %d are different.\n", i);
		if (abs(x[i] - hostParticles[base + 0]) > diffTol) printf("X-coordinates of particle %d are too(?) different.\n", i);
		if (abs(y[i] - hostParticles[base + 1]) > diffTol) printf("Y-coordinates of particle %d are too(?) different.", i);
		if (abs(z[i] - hostParticles[base + 2]) > diffTol) printf("Z-coordinates of particle %d are too(?) different.", i);

	}*/
	
	double t2 = omp_get_wtime(); // in seconds

	// CUDA time measurement
	hipEventRecord(stop); // stop recording

	hipEventSynchronize(stop); // synchronized stop
	float miliseconds = 0; // init time 
	hipEventElapsedTime(&miliseconds, start, stop); // count time

	#ifdef LOGGING
	printf("Time: %f seconds\n",(t2-t1));
	#endif

	// times should be almost equal (both measures almost the same)
	printf("OpenMP: %d %f\n", threads, (t2-t1));
	printf("CUDA: %f\n", miliseconds/(1000.0));
	
	// free CPU arrays
#ifdef WITH_CPU
	delete [] xnew;
	delete [] ynew;
	delete [] znew;
#endif

	delete[] hostParticles;
	delete[] hostVelocities;

	// free GPU arrays
	hipFree(devParticles);
	hipFree(devParticlesNew);
	hipFree(devVelocities);

	return 0;
}

void recycleBin()
{
#ifdef ABCDEFGH
	// pointers to arrays on GPU
	float * devX, *devY, *devZ;
	float * devXnew, *devYnew, *devZnew;
	float * devM;
	float * devVx, *devVy, *devVz;
	// allocation of arrays for XYZ coordinates
	HANDLE_ERROR(hipMalloc((void**)&devX, sconf.amount * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&devY, sconf.amount * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&devZ, sconf.amount * sizeof(float)));

	HANDLE_ERROR(hipMalloc((void**)&devM, sconf.amount * sizeof(float)));

	HANDLE_ERROR(hipMalloc((void**)&devXnew, sconf.amount * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&devYnew, sconf.amount * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&devZnew, sconf.amount * sizeof(float)));

	HANDLE_ERROR(hipMalloc((void**)&devVx, sconf.amount * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&devVy, sconf.amount * sizeof(float)));
	HANDLE_ERROR(hipMalloc((void**)&devVz, sconf.amount * sizeof(float)));

	// copy CPU -> GPU
	// hipMemcpy(to, from, amount, type)
	// coordinates
	HANDLE_ERROR(hipMemcpy(devX, x, sconf.amount, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devY, y, sconf.amount, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devZ, z, sconf.amount, hipMemcpyHostToDevice));
	// masses - maybe to constant/read-only memory?
	HANDLE_ERROR(hipMemcpy(devM, m, sconf.amount, hipMemcpyHostToDevice));
	// new coordinates
	HANDLE_ERROR(hipMemcpy(devXnew, xnew, sconf.amount, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devYnew, ynew, sconf.amount, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devZnew, znew, sconf.amount, hipMemcpyHostToDevice));
	// velocities
	HANDLE_ERROR(hipMemcpy(devVx, vx, sconf.amount, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devVy, vy, sconf.amount, hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(devVz, vz, sconf.amount, hipMemcpyHostToDevice));



	// free GPU arrays
	hipFree(devX);
	hipFree(devY);
	hipFree(devZ);

	hipFree(devM);

	hipFree(devXnew);
	hipFree(devYnew);
	hipFree(devZnew);

	hipFree(devVx);
	hipFree(devVy);
	hipFree(devVz);
#endif
}


void getDeviceInfo()
{
	int devCount = 0;
	hipGetDeviceCount(&devCount);

	printf("Amount of devices: %d\n", devCount);

	for (int i = 0; i < devCount; i++) {
		hipDeviceProp_t prop;
		HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
		/*if (cudaStatus != hipSuccess)
		{
		printf("%s\n", hipGetErrorString(cudaStatus));
		continue;
		}*/
		printf("Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);

		printf("  Multiprocessor count: %d\n", prop.multiProcessorCount);

		printf("  Total global Memory: %d\n", prop.totalGlobalMem);

		printf("  Total const memory: %d\n", prop.totalConstMem);

		printf("  Shared memory per block: %d\n", prop.sharedMemPerBlock);

		printf("  Shared memory per multiprocessor: %d\n", prop.sharedMemPerMultiprocessor);

		printf("  Max threads per block: %d\n", prop.maxThreadsPerBlock);

		printf("  Max threads per multiprocessor: %d\n", prop.maxThreadsPerMultiProcessor);

		size_t size;
		hipDeviceGetLimit(&size, hipLimitMallocHeapSize);
		printf("Heap size limit: %d\n", size);

		printf("  Memory Clock Rate (KHz): %d\n",
			prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n",
			prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
			2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
	}
}
